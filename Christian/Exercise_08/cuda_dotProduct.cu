#include "hip/hip_runtime.h"
#include "timer.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <vector>
#include <iostream>

__global__ void dot_product(int* x, int* y, int* dot, int N) {

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ int cache[128];

    int temp = 0;
    while (index < N) {
        temp += (x[index] + y[index]) * (x[index] - y[index]);
        index += stride;
    }

    cache[threadIdx.x] = temp;

    __syncthreads();

    for (int i = blockDim.x/2; i > 0; i/= 2) {
        __syncthreads();
        if (threadIdx.x < i)
            cache[threadIdx.x] += cache[threadIdx.x + i];
    }

    if (threadIdx.x == 0)
        atomicAdd(dot, cache[0]);

}


int main() {

    Timer timer;
    int N = 1000;

    int *x = (int *)malloc(sizeof(int) * N);
    int *y = (int *)malloc(sizeof(int) * N);
    int *dot = (int *)malloc(sizeof(int));

    for (int i = 0; i < N; i++) {
        x[i] = 1;
        y[i] = 2;
    }
    *dot = 0;

    int *cuda_x;
    int *cuda_y;
    int *cuda_dot;
    hipMalloc(&cuda_x, sizeof(int) * N);
    hipMalloc(&cuda_y, sizeof(int) * N);
    hipMalloc(&cuda_dot, sizeof(int));

    hipMemcpy(cuda_x, x, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_y, y, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_dot, dot, sizeof(int), hipMemcpyHostToDevice);

    std::vector<double> timings;
    for(int reps=0; reps < 10; ++reps) {
        timer.reset();
        dot_product<<<N/256, 128>>>(cuda_x, cuda_y, cuda_dot, N);
        hipMemcpy(dot, cuda_dot, sizeof(int), hipMemcpyDeviceToHost);
        timings.push_back(timer.get());        
        std::cout << "Dot Product = " << *dot << std::endl;
        *dot = 0;
        hipMemcpy(cuda_dot, dot, sizeof(int), hipMemcpyHostToDevice);
    }

    std::sort(timings.begin(), timings.end());
    double time_elapsed = timings[10/2];

    std::cout << "Time elapsed: " << time_elapsed << std::endl << std::endl;

    

    return EXIT_SUCCESS;
}